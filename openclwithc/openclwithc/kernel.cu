#include "hip/hip_runtime.h"
#include "main.h"

/*
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
*/

__global__ void matrixKernel(long width, long height, long **result, long **Matrix, long **weight) {
	int range = MASK_RANGE / 2;
	int nowx = blockIdx.y * blockDim.y + threadIdx.y;
	int nowy = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (nowx > height - 1 || nowy > width) return;

	for (int i = -range; i < range + 1; i++) {
		for (int j = -range; j < range + 1; j++) {
			result[nowx][nowy] += (weight[i + range][j + range] * Matrix[nowx + i][nowy + j]);
		}
	}

}

// Helper function for using CUDA to add vectors in parallel.
//hipError_t calWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//	int *dev_a = 0;
//	int *dev_b = 0;
//	int *dev_c = 0;
//	hipError_t cudaStatus;
//
//	// Choose which GPU to run on, change this on a multi-GPU system.
//	cudaStatus = hipSetDevice(0);
//	CUDA_CHECK_ERROR(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
//
//	// Allocate GPU buffers for three vectors (two input, one output).
//	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//	CUDA_CHECK_ERROR(cudaStatus, "hipMalloc failed!\n");
//
//	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//	CUDA_CHECK_ERROR(cudaStatus, "hipMalloc failed!\n");
//
//	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//	CUDA_CHECK_ERROR(cudaStatus, "hipMalloc failed!\n");
//
//	// Copy input vectors from host memory to GPU buffers.
//	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//	CUDA_CHECK_ERROR(cudaStatus, "hipMemcpy failed!\n");
//
//	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//	CUDA_CHECK_ERROR(cudaStatus, "hipMemcpy failed!\n");
//
//	// Launch a kernel on the GPU with one thread for each element.
//	addKernel <<<1, size >>>(dev_c, dev_a, dev_b);
//
//	// Check for any errors launching the kernel
//	cudaStatus = hipGetLastError();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//		goto Error;
//	}
//
//	// hipDeviceSynchronize waits for the kernel to finish, and returns
//	// any errors encountered during the launch.
//	cudaStatus = hipDeviceSynchronize();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//		goto Error;
//	}
//
//	// Copy output vector from GPU buffer to host memory.
//	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//	CUDA_CHECK_ERROR(cudaStatus, "hipMemcpy failed!\n");
//
//Error:
//	hipFree(dev_c);
//	hipFree(dev_a);
//	hipFree(dev_b);
//
//	return cudaStatus;
//}

// Helper function for using CUDA to add vectors in parallel.
hipError_t matrixWithCuda(long **c, long **a, long b[MASK_RANGE][MASK_RANGE], unsigned int width, unsigned int height)
{
	long **dev_a;
	long **buff_a = (long **)malloc(height * sizeof(long *));
	long **dev_b;
	long **buff_b = (long **)malloc(MASK_RANGE * sizeof(long *));
	long **dev_c;
	long **buff_c = (long **)malloc(height * sizeof(long *));
	hipError_t cudaStatus;

	memset(buff_a, NULL, sizeof(long *) * height);
	memset(buff_b, NULL, sizeof(long *) * MASK_RANGE);
	memset(buff_c, NULL, sizeof(long *) * height);

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	CUDA_CHECK_ERROR(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");

	// Allocate GPU buffers for height Areea.
	LOG_MSG("Allocate GPU buffer dev c");
	cudaStatus = hipMalloc((void***)&dev_c, sizeof(long *) * height);
	CUDA_CHECK_ERROR(cudaStatus, "hipMalloc failed!\n");

	cudaStatus = hipMemset(dev_c, NULL, height * sizeof(long *));
	CUDA_CHECK_ERROR(cudaStatus, "memory initialize Error");

	LOG_MSG("Allocate GPU buffer dev a");
	cudaStatus = hipMalloc((void***)&dev_a, sizeof(long *) * height);
	CUDA_CHECK_ERROR(cudaStatus, "hipMalloc failed!\n");

	cudaStatus = hipMemset(dev_a, NULL, height * sizeof(long *));
	CUDA_CHECK_ERROR(cudaStatus, "memory initialize Error");


	LOG_MSG("Allocate GPU buffer dev b");
	cudaStatus = hipMalloc((void***)&dev_b, sizeof(long *) * MASK_RANGE);
	CUDA_CHECK_ERROR(cudaStatus, "hipMalloc failed!\n");

	cudaStatus = hipMemset(dev_b, NULL, MASK_RANGE * sizeof(long *));
	CUDA_CHECK_ERROR(cudaStatus, "memory initialize Error");


	LOG_MSG("Allocate GPU buffer specific dev b and memcpy b to dev b");
	for (int rp = 0; rp < MASK_RANGE; rp++) {
		// Allocate GPU buffers for Specific Area.
		cudaStatus = hipMalloc((void**)&(buff_b[rp]), MASK_RANGE * sizeof(long));
		CUDA_CHECK_ERROR(cudaStatus, "hipMalloc failed!\n");

		cudaStatus = hipMemset(buff_b[rp], 0, MASK_RANGE * sizeof(long));
		CUDA_CHECK_ERROR(cudaStatus, "memory initialize Error");

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(buff_b[rp], b[rp], MASK_RANGE *sizeof(long), hipMemcpyHostToDevice);
		CUDA_CHECK_ERROR(cudaStatus, "hipMemcpy failed!\n");
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_b, buff_b, MASK_RANGE * sizeof(long *), hipMemcpyHostToDevice);
	CUDA_CHECK_ERROR(cudaStatus, "hipMemcpy failed!\n");

	for (unsigned int rp = 0; rp < height; rp++) {
		LOG_MSG("Allocate buffer specific dev c ");

		// Allocate GPU buffers for Specific Area.
		cudaStatus = hipMalloc((void**)&(buff_c[rp]), sizeof(long) * width);
		CUDA_CHECK_ERROR(cudaStatus, "hipMalloc failed!\n");

		cudaStatus = hipMemset(buff_c[rp], 0, width * sizeof(long));
		CUDA_CHECK_ERROR(cudaStatus, "memory initialize Error");
		
		LOG_MSG("Memory copy c to specific buffer c ");
		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(buff_c[rp], c[rp], sizeof(long) * width, hipMemcpyHostToDevice);
		CUDA_CHECK_ERROR(cudaStatus, "hipMemcpy failed!\n");



		LOG_MSG("Allocate buffer specific dev a");
		// Allocate GPU buffers for Specific Area.
		cudaStatus = hipMalloc((void**)&(buff_a[rp]), sizeof(long) * width);
		CUDA_CHECK_ERROR(cudaStatus, "hipMalloc failed!\n");

		cudaStatus = hipMemset(buff_a[rp], 0, width * sizeof(long));
		CUDA_CHECK_ERROR(cudaStatus, "memory initialize Error");

		LOG_MSG("Memory copy c to buffer c ");
		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(buff_a[rp], a[rp], sizeof(long) * width, hipMemcpyHostToDevice);
		CUDA_CHECK_ERROR(cudaStatus, "hipMemcpy failed!\n");
	}

	LOG_MSG("Memory copy buff c to  dev c ");
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_c, buff_c, sizeof(long *) * height, hipMemcpyHostToDevice);
	CUDA_CHECK_ERROR(cudaStatus, "hipMemcpy failed!\n");

	LOG_MSG("Memory copy buff a to dev a ");
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, buff_a, sizeof(long *) * height, hipMemcpyHostToDevice);
	CUDA_CHECK_ERROR(cudaStatus, "hipMemcpy failed!\n");

	LOG_MSG("Calculating with CUDA Kernel start");
	// Launch a kernel on the GPU with one thread for each element.
	matrixKernel <<< 1, width*height >>>(width, height, dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "matrixKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching matrixKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.

	cudaStatus = hipMemcpy(buff_c, dev_c, height * sizeof(long*), hipMemcpyDeviceToHost);
	CUDA_CHECK_ERROR(cudaStatus, "hipMemcpy failed!\n");

	for (unsigned int rp = 0; rp < height; rp++) {
		cudaStatus = hipMemcpy(c[rp], buff_c[rp], width * sizeof(long), hipMemcpyDeviceToHost);
		CUDA_CHECK_ERROR(cudaStatus, "hipMemcpy failed!\n");
	}

Error:
	free(buff_a);
	free(buff_c);
	free(buff_b);
	hipFree(dev_b);
	hipFree(dev_a);
	hipFree(dev_c);

	return cudaStatus;
}